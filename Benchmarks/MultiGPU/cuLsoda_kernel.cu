#include "hip/hip_runtime.h"
/*
 *  cuLsoda_kernel.cu
 *  cuLsoda
 *
 */
 #ifndef _CULSODA_CU_H_
 #define _CULSODA_CU_H_
 
 #include "cuLsoda.cu.h"
 
 #define REAL double
 
 template<typename Fex, typename Jex>
__global__ void Sanders(Fex fex, int *neq, REAL *y, REAL *t, REAL *tout, int *itol, REAL *rtol, REAL *atol, int *itask, int *istate, int *iopt, REAL *rwork, int *lrw, int *iwork, int *liw, Jex jac, int *jt, struct cuLsodaCommonBlock *common, int *err, int probSize)
{
	int me = threadIdx.x + blockIdx.x * blockDim.x;
//	printf("Thread ID: %d\tProbsize: %d\n",me,probSize);
	if(me < probSize){
//	printf("neq: %d\ty[0]: %f\ty[1]: %f\ty[2]: %f\ty[3]: %f\tt: %f\ttout: %f\n",neq[me],y[4*me],y[4*me+1],y[4*me+2],y[4*me+3],t[me],tout[me]);
	err[me] = dlsoda_(fex, &neq[me], &y[4*me], &t[me], &tout[me], &itol[me], &rtol[me], &atol[me], &itask[me], &istate[me], &iopt[me], &rwork[86*me], &lrw[me], &iwork[24*me], &liw[me], jac, &jt[me], &common[me]);
	
	}
	__syncthreads();
}


#endif

